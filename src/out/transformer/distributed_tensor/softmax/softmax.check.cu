#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif
#include "cudnn_header.h"
#include "nccl_header.h"
#include <string.h>
#include <cblas.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include "cublas_header.h"
#include <stdbool.h>
#include "mpi_header.h"
#include "scanner_header.h"
/************* Functions **************/
__global__ void x11(float* x12, float x13, int x14) {
  // begin generating kernel function for FILL of type Float
  int x15 = gridDim.x * blockDim.x;
  int x16 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x16 < x14) {
    x12[x16] = x13;
    x16 = x16 + x15;
  }
  // end generating kernel function for FILL of type Float
}
__global__ void x24(float* x25, float* x26, float* x27, int x28) {
  // begin generating kernel function for ADD of type Float
  int x29 = gridDim.x * blockDim.x;
  int x30 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x30 < x28) {
    int x31 = x30;
    x27[x31] = x25[x31] + x26[x31];
    x30 = x30 + x29;
  }
  // end generating kernel function for ADD of type Float
}
__global__ void x36(float* x37, float* x38, int x39) {
  // begin generating kernel function for ACCUM of type Float
  int x40 = gridDim.x * blockDim.x;
  int x41 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x41 < x39) {
    int x42 = x41;
    x37[x42] = x37[x42] + x38[x42];
    x41 = x41 + x40;
  }
  // end generating kernel function for ACCUM of type Float
}
/**************** Snippet ****************/
void Snippet(int x0) {
  // begin setting up the MPI/NCCL environment
  int x1 = 0;
  int x2 = 0;
  MPICHECK(MPI_Init(NULL, NULL));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &x2));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &x1));
  MPICHECK(MPI_Barrier(MPI_COMM_WORLD));
  CUDA_CALL(hipSetDevice(x2));
  ncclUniqueId x3;
  NCCLCHECK(ncclGetUniqueId(&x3));
  MPICHECK(MPI_Bcast(&x3, NCCL_UNIQUE_ID_BYTES, MPI_CHAR, 0, MPI_COMM_WORLD));
  ncclComm_t x4;
  NCCLCHECK(ncclCommInitRank(&x4, x1, x3, x2));
  hipStream_t x5;
  CUDA_CALL(hipStreamCreateWithFlags(&x5, hipStreamNonBlocking));
  int x6 = x2;
  // end setting up the MPI/NCCL environment
  // begin setting up the CUDNN environment
  hipdnnHandle_t x7;
  CUDNNCHECK(hipdnnCreate(&x7));
  // end setting up the CUDNN environment
  // begin initializing GPU array of size 9 and type Float at device (pre-rename) x39 from binary file weight
  float* x8 = (float*)malloc(9 * sizeof(float));
  CUDA_CALL(hipSetDevice(x6));
  float* x9 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x9, (size_t)(9 * sizeof(float))));
  scan_float_rank("golden/weight", x6, x8, 9);
  CUDA_CALL(hipMemcpy(x9, x8, (size_t)(9 * sizeof(float)), hipMemcpyHostToDevice));
  // end initializing GPU array of size 9 and type Float at device (pre-rename) x39 from binary file weight
  // begin initializing fixed GPU array of size 9 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x6));
  float* x10 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x10, (size_t)(9 * sizeof(float))));
  x11<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x10, 0, 9);
  // end initializing fixed GPU array of size 9 and type Float and device (pre-rename) x39
  // begin initializing GPU array of size 9 and type Float at device (pre-rename) x39 from binary file input
  float* x17 = (float*)malloc(9 * sizeof(float));
  CUDA_CALL(hipSetDevice(x6));
  float* x18 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x18, (size_t)(9 * sizeof(float))));
  scan_float_rank("golden/input", x6, x17, 9);
  CUDA_CALL(hipMemcpy(x18, x17, (size_t)(9 * sizeof(float)), hipMemcpyHostToDevice));
  // end initializing GPU array of size 9 and type Float at device (pre-rename) x39 from binary file input
  // begin creating and setting tensor descriptor of shape List(2, 1, 3, 3)
  hipdnnTensorDescriptor_t x19;
  CUDNNCHECK(hipdnnCreateTensorDescriptor(&x19));
  CUDNNCHECK(hipdnnSetTensor4dDescriptor(x19, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 2, 1, 3, 3));
  // end creating and setting tensor descriptor
  // begin allocating gpu array for the output of softmax
  CUDA_CALL(hipSetDevice(x6));
  float* x20 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x20, (size_t)(18 * sizeof(float))));
  // end allocating gpu array for the output of softmax
  // begin softmax forward pass
  float x21 = 1.0;
  float x22 = 0.0;
  CUDNNCHECK(hipdnnSoftmaxForward(x7, HIPDNN_SOFTMAX_FAST, HIPDNN_SOFTMAX_MODE_INSTANCE, &x21, x19, x9, &x22, x19, x20));
  // end softmax forward pass
  // begin computing ADD on GPU for size 9 and type Float at device (pre-rename) x39 with left_operand x108 and right_operand x134
  CUDA_CALL(hipSetDevice(x6));
  float* x23 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x23, (size_t)(9 * sizeof(float))));
  x24<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x18, x20, x23, 9);
  // end computing ADD on GPU for size 9 and type Float at device (pre-rename) x39 with left_operand x108 and right_operand x134
  // begin initializing fixed GPU array of size 9 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x6));
  float* x32 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x32, (size_t)(9 * sizeof(float))));
  x11<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x32, 0, 9);
  // end initializing fixed GPU array of size 9 and type Float and device (pre-rename) x39
  // begin initializing fixed GPU array of size 9 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x6));
  float* x33 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x33, (size_t)(9 * sizeof(float))));
  x11<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x33, 0, 9);
  // end initializing fixed GPU array of size 9 and type Float and device (pre-rename) x39
  // begin checking GPU array of size 9 and type Float at device (pre-name) x39 again binary file loss
  float* x34 = (float*)malloc(9 * sizeof(float));
  CUDA_CALL(hipMemcpy(x34, x23, (size_t)(9 * sizeof(float)), hipMemcpyDeviceToHost));
  check_float_array_rank("golden/loss", x6, (float*)malloc(9 * sizeof(float)), x34, 9);
  // end checking GPU array of size 9 and type Float at device (pre-name) x39 again binary file loss
  // begin initializing fixed GPU array of size 9 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x6));
  float* x35 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x35, (size_t)(9 * sizeof(float))));
  x11<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x35, 1, 9);
  // end initializing fixed GPU array of size 9 and type Float and device (pre-rename) x39
  // begin computing ACCUM on GPU for size 9 and type Float at device (pre-rename) x39 with base_operand x209 and addition_operand x233
  CUDA_CALL(hipSetDevice(x6));
  x36<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x33, x35, 9);
  // end computing ACCUM on GPU for size 9 and type Float at device (pre-rename) x39 with base_operand x209 and addition_operand x233
  // begin computing ACCUM on GPU for size 9 and type Float at device (pre-rename) x39 with base_operand x196 and addition_operand x233
  CUDA_CALL(hipSetDevice(x6));
  x36<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x32, x35, 9);
  // end computing ACCUM on GPU for size 9 and type Float at device (pre-rename) x39 with base_operand x196 and addition_operand x233
  // begin allocating gpu array for the gradient of input of softmax
  CUDA_CALL(hipSetDevice(x6));
  float* x43 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x43, (size_t)(18 * sizeof(float))));
  // end allocating gpu array for the gradient of input of softmax
  // begin softmax backward pass
  float x44 = 1.0;
  float x45 = 0.0;
  CUDNNCHECK(hipdnnSoftmaxBackward(x7, HIPDNN_SOFTMAX_FAST, HIPDNN_SOFTMAX_MODE_INSTANCE, &x44, x19, x20, x19, x33, &x45, x19, x43));
  // end softmax backward pass
  // begin computing ACCUM on GPU for size 9 and type Float at device (pre-rename) x39 with base_operand x67 and addition_operand x290
  CUDA_CALL(hipSetDevice(x6));
  x36<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x10, x43, 9);
  // end computing ACCUM on GPU for size 9 and type Float at device (pre-rename) x39 with base_operand x67 and addition_operand x290
  // begin checking GPU array of size 9 and type Float at device (pre-name) x39 again binary file weight_grad
  float* x46 = (float*)malloc(9 * sizeof(float));
  CUDA_CALL(hipMemcpy(x46, x10, (size_t)(9 * sizeof(float)), hipMemcpyDeviceToHost));
  check_float_array_rank("golden/weight_grad", x6, (float*)malloc(9 * sizeof(float)), x46, 9);
  // end checking GPU array of size 9 and type Float at device (pre-name) x39 again binary file weight_grad
  // begin checking GPU array of size 9 and type Float at device (pre-name) x39 again binary file input_grad
  float* x47 = (float*)malloc(9 * sizeof(float));
  CUDA_CALL(hipMemcpy(x47, x32, (size_t)(9 * sizeof(float)), hipMemcpyDeviceToHost));
  check_float_array_rank("golden/input_grad", x6, (float*)malloc(9 * sizeof(float)), x47, 9);
  // end checking GPU array of size 9 and type Float at device (pre-name) x39 again binary file input_grad
  hipdnnDestroyTensorDescriptor(x19);
  CUDNNCHECK(hipdnnDestroy(x7));
  MPICHECK(MPI_Finalize());
  NCCLCHECK(ncclCommDestroy(x4));
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
