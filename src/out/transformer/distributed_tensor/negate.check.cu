#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#include "nccl_header.h"
#include <string.h>
#include <cblas.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include "mpi_header.h"
/************* Functions **************/
__global__ void x12(float* x13, float x14, int x15) {
  int x16 = gridDim.x * blockDim.x;
  int x17 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x17 < x15) {
    x13[x17] = x14;
    x17 = x17 + x16;
  }
}
__global__ void x20(float* x21, float* x22, int x23) {
  // begin generating kernel function for ACCUM of type Float
  int x24 = gridDim.x * blockDim.x;
  int x25 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x25 < x23) {
    int x26 = x25;
    x21[x26] = x21[x26] + x22[x26];
    x25 = x25 + x24;
  }
  // end generating kernel function for ACCUM of type Float
}
__global__ void x27(float* x28, float* x29, int x30) {
  // begin generating kernel function for NEGATE of type Float
  int x31 = gridDim.x * blockDim.x;
  int x32 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x32 < x30) {
    int x33 = x32;
    x29[x33] = 0.0 - x28[x33];
    x32 = x32 + x31;
  }
  // end generating kernel function for NEGATE of type Float
}
__global__ void x34(float* x35, float* x36, float* x37, int x38) {
  // begin generating kernel function for SGD of type Float
  int x39 = gridDim.x * blockDim.x;
  int x40 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x40 < x38) {
    int x41 = x40;
    float x42 = x37[x41] * 0.5 + x36[x41];
    x35[x41] = x35[x41] - x42 * 1.0E-4;
    x37[x41] = x42;
    x40 = x40 + x39;
  }
  // end generating kernel function for SGD of type Float
}
/**************** Snippet ****************/
void Snippet(int x0) {
  // begin setting up the MPI/NCCL environment
  int x1 = 0;
  int x2 = 0;
  MPICHECK(MPI_Init(NULL, NULL));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &x2));
  int x3 = MPI_Comm_size(MPI_COMM_WORLD, &x1);
  MPICHECK(x3);
  MPICHECK(MPI_Barrier(MPI_COMM_WORLD));
  CUDA_CALL(hipSetDevice(x2));
  ncclUniqueId x4;
  NCCLCHECK(ncclGetUniqueId(&x4));
  MPICHECK(MPI_Bcast(&x4, NCCL_UNIQUE_ID_BYTES, MPI_CHAR, 0, MPI_COMM_WORLD));
  ncclComm_t x5;
  NCCLCHECK(ncclCommInitRank(&x5, x1, x4, x2));
  hipStream_t x6;
  CUDA_CALL(hipStreamCreateWithFlags(&x6, hipStreamNonBlocking));
  int x7 = x2;
  // end setting up the MPI/NCCL environment
  // begin initializing random GPU array of size 512 and type Float at device (pre-rename) x39
  float* x8 = (float*)malloc(512 * sizeof(float));
  int x9 = 0;
  while (x9 != 512) {
    x8[x9] = (float)(rand() - RAND_MAX / 2) / (float)RAND_MAX;
    x9 = x9 + 1;
  }
  CUDA_CALL(hipSetDevice(x7));
  float* x10 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x10, (size_t)(512 * sizeof(float))));
  CUDA_CALL(hipMemcpy(x10, x8, (size_t)(512 * sizeof(float)), hipMemcpyHostToDevice));
  // end initializing random GPU array of size 512 and type Float at device (pre-rename) x39
  // begin initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x7));
  float* x11 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x11, (size_t)(512 * sizeof(float))));
  x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x11, 0, 512);
  // end initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  // begin initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x7));
  float* x18 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x18, (size_t)(512 * sizeof(float))));
  x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x18, 0, 512);
  // end initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  int x19 = 0;
  while (x19 != 5) {
    // begin initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
    CUDA_CALL(hipSetDevice(x7));
    float* x43 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x43, (size_t)(512 * sizeof(float))));
    x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x43, 0, 512);
    // end initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
    // begin initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
    CUDA_CALL(hipSetDevice(x7));
    float* x44 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x44, (size_t)(512 * sizeof(float))));
    x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x44, 1, 512);
    // end initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
    // begin computing ACCUM on GPU for size 512 and type Float at device (pre-rename) x39 with base_operand x137 and addition_operand x150
    CUDA_CALL(hipSetDevice(x7));
    x20<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x43, x44, 512);
    // end computing ACCUM on GPU for size 512 and type Float at device (pre-rename) x39 with base_operand x137 and addition_operand x150
    // begin computing NEG on GPU for size 512 and type Float at device (pre-rename) x39 with operand x137
    CUDA_CALL(hipSetDevice(x7));
    float* x45 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x45, (size_t)(512 * sizeof(float))));
    x27<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x43, x45, 512);
    // end computing NEG on GPU for size 512 and type Float at device (pre-rename) x39 with operand x137
    // begin computing ACCUM on GPU for size 512 and type Float at device (pre-rename) x39 with base_operand x81 and addition_operand x200
    CUDA_CALL(hipSetDevice(x7));
    x20<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x11, x45, 512);
    // end computing ACCUM on GPU for size 512 and type Float at device (pre-rename) x39 with base_operand x81 and addition_operand x200
    // begin computing SGD on GPU for size 512 and type Float at device (pre-name) x39 with weight x65, grad x81, and momentum x119
    CUDA_CALL(hipSetDevice(x7));
    x34<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x10, x11, x18, 512);
    // end computing SGD on GPU for size 512 and type Float at device (pre-name) x39 with weight x65, grad x81, and momentum x119
    x19 = x19 + 1;
  }
  // Only declare recv buffer if this is the root
  bool x46 = x7 == 0;
  float* x47 = x46 ? ({
    float* x48 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x48, (size_t)(1024 * sizeof(float))));
    x48;
  }) : ({
    float* x49 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x49, (size_t)0));
    x49;
  });
  // Gather by groups of NCCL send/recv
  NCCLCHECK(ncclGroupStart());
  lms.thirdparty.nccltypeless$nccl_result x50 = ncclSend(x10, (size_t)1024, ncclFloat32, 0, x5, x6);
  NCCLCHECK(x50);
  if (x46) {
    int x51 = x1;
    int x52 = 0;
    while (x52 != x51) {
      int x53 = x52;
      NCCLCHECK(ncclRecv(x47 + x53 * 512, (size_t)1024, ncclFloat32, x53, x5, x6));
      x52 = x52 + 1;
    }
  }
  NCCLCHECK(ncclGroupEnd());
  // print the array only if this is the root
  if (x46) {
    // begin copying GPU array x308 to CPU and print for size 1024 and type Float
    float* x54 = (float*)malloc(1024 * sizeof(float));
    CUDA_CALL(hipMemcpy(x54, x47, (size_t)(1024 * sizeof(float)), hipMemcpyDeviceToHost));
    int x55 = 0;
    while (x55 != 1024) {
      printf("%f ", x54[x55]);
      x55 = x55 + 1;
    }
    printf("\n");
    // end copying GPU array x308 to CPU and print for size 1024 and type Float
  }
  printf("compile");
  MPICHECK(MPI_Finalize());
  NCCLCHECK(ncclCommDestroy(x5));
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
