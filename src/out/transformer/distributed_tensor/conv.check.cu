#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#include "cudnn_header.h"
#include "nccl_header.h"
#include <string.h>
#include <cblas.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include "mpi_header.h"
/************* Functions **************/
__global__ void x12(float* x13, float x14, int x15) {
  int x16 = gridDim.x * blockDim.x;
  int x17 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x17 < x15) {
    x13[x17] = x14;
    x17 = x17 + x16;
  }
}
__global__ void x20(float* x21, float* x22, int x23) {
  // begin generating kernel function for ACCUM of type Float
  int x24 = gridDim.x * blockDim.x;
  int x25 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x25 < x23) {
    int x26 = x25;
    x21[x26] = x21[x26] + x22[x26];
    x25 = x25 + x24;
  }
  // end generating kernel function for ACCUM of type Float
}
__global__ void x27(float* x28, float* x29, float* x30, int x31) {
  // begin generating kernel function for SGD of type Float
  int x32 = gridDim.x * blockDim.x;
  int x33 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x33 < x31) {
    int x34 = x33;
    float x35 = x30[x34] * 0.5 + x29[x34];
    x28[x34] = x28[x34] - x35 * 1.0E-4;
    x30[x34] = x35;
    x33 = x33 + x32;
  }
  // end generating kernel function for SGD of type Float
}
/**************** Snippet ****************/
void Snippet(int x0) {
  // begin setting up the MPI/NCCL environment
  int x1 = 0;
  int x2 = 0;
  MPICHECK(MPI_Init(NULL, NULL));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &x2));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &x1));
  MPICHECK(MPI_Barrier(MPI_COMM_WORLD));
  CUDA_CALL(hipSetDevice(x2));
  ncclUniqueId x3;
  NCCLCHECK(ncclGetUniqueId(&x3));
  MPICHECK(MPI_Bcast(&x3, NCCL_UNIQUE_ID_BYTES, MPI_CHAR, 0, MPI_COMM_WORLD));
  ncclComm_t x4;
  NCCLCHECK(ncclCommInitRank(&x4, x1, x3, x2));
  hipStream_t x5;
  CUDA_CALL(hipStreamCreateWithFlags(&x5, hipStreamNonBlocking));
  int x6 = x2;
  // end setting up the MPI/NCCL environment
  // begin setting up the CUDNN environment
  hipdnnHandle_t x7;
  CUDNNCHECK(hipdnnCreate(&x7));
  // end setting up the CUDNN environment
  // begin initializing random GPU array of size 18 and type Float at device (pre-rename) x39
  float* x8 = (float*)malloc(18 * sizeof(float));
  int x9 = 0;
  while (x9 != 18) {
    x8[x9] = (float)(rand() - RAND_MAX / 2) / (float)RAND_MAX;
    x9 = x9 + 1;
  }
  CUDA_CALL(hipSetDevice(x6));
  float* x10 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x10, (size_t)(18 * sizeof(float))));
  CUDA_CALL(hipMemcpy(x10, x8, (size_t)(18 * sizeof(float)), hipMemcpyHostToDevice));
  // end initializing random GPU array of size 18 and type Float at device (pre-rename) x39
  NCCLCHECK(ncclAllReduce(x10, x10, (size_t)(18 * sizeof(float)), ncclFloat32, ncclSum, x4, x5));
  // begin initializing fixed GPU array of size 18 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x6));
  float* x11 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x11, (size_t)(18 * sizeof(float))));
  x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x11, 0, 18);
  // end initializing fixed GPU array of size 18 and type Float and device (pre-rename) x39
  // begin initializing fixed GPU array of size 18 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x6));
  float* x18 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x18, (size_t)(18 * sizeof(float))));
  x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x18, 0, 18);
  // end initializing fixed GPU array of size 18 and type Float and device (pre-rename) x39
  int x19 = 0;
  while (x19 != 10) {
    // begin initializing random GPU array of size 81 and type Float at device (pre-rename) x39
    float* x36 = (float*)malloc(81 * sizeof(float));
    int x37 = 0;
    while (x37 != 81) {
      x36[x37] = (float)(rand() - RAND_MAX / 2) / (float)RAND_MAX;
      x37 = x37 + 1;
    }
    CUDA_CALL(hipSetDevice(x6));
    float* x38 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x38, (size_t)(81 * sizeof(float))));
    CUDA_CALL(hipMemcpy(x38, x36, (size_t)(81 * sizeof(float)), hipMemcpyHostToDevice));
    // end initializing random GPU array of size 81 and type Float at device (pre-rename) x39
    // begin initializing fixed GPU array of size 162 and type Float and device (pre-rename) x39
    CUDA_CALL(hipSetDevice(x6));
    float* x39 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x39, (size_t)(162 * sizeof(float))));
    x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x39, 1, 162);
    // end initializing fixed GPU array of size 162 and type Float and device (pre-rename) x39
    // begin creating and setting tensor descriptor of shape List(2, 1, 9, 9)
    hipdnnTensorDescriptor_t x40;
    CUDNNCHECK(hipdnnCreateTensorDescriptor(&x40));
    CUDNNCHECK(hipdnnSetTensor4dDescriptor(x40, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 2, 1, 9, 9));
    // end creating and setting tensor descriptor
    // begin creating and setting tensor descriptor of shape List(2, 2, 9, 9)
    hipdnnTensorDescriptor_t x41;
    CUDNNCHECK(hipdnnCreateTensorDescriptor(&x41));
    CUDNNCHECK(hipdnnSetTensor4dDescriptor(x41, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 2, 2, 9, 9));
    // end creating and setting tensor descriptor
    // begin creating and setting filter descriptor of shape List(2, 1, 3, 3)
    hipdnnFilterDescriptor_t x42;
    CUDNNCHECK(hipdnnCreateFilterDescriptor(&x42));
    CUDNNCHECK(hipdnnSetFilter4dDescriptor(x42, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 2, 1, 3, 3));
    // end creating and setting filter descriptor
    // begin creating and setting convolution descriptor of padding: List(1, 1), strides: List(1, 1), dilation: List(1, 1)
    hipdnnConvolutionDescriptor_t x43;
    CUDNNCHECK(hipdnnCreateConvolutionDescriptor(&x43));
    CUDNNCHECK(hipdnnSetConvolution2dDescriptor(x43, 1, 1, 1, 1, 1, 1, HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT));
    // end creating and setting convolution descriptor
    // begin allocating gpu array for the gradient of filter of convolution
    CUDA_CALL(hipSetDevice(x6));
    float* x44 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x44, (size_t)(18 * sizeof(float))));
    // end allocating gpu array for the gradient of filter of convolution
    // begin finding convolution backward filter algorithm
    hipdnnConvolutionBwdFilterAlgoPerf_t x45;
    int x46 = 0;
    CUDNNCHECK(hipdnnFindConvolutionBackwardFilterAlgorithm(x7, x40, x41, x43, x42, 1, &x46, &x45));
    hipdnnConvolutionBwdFilterAlgo_t x47 = x45.algo;
    // end finding convolution backward filter algorithm
    // begin finding convolution backward filter workspace size
    size_t x48 = (size_t)0;
    CUDNNCHECK(hipdnnGetConvolutionBackwardFilterWorkspaceSize(x7, x40, x41, x43, x42, x47, &x48));
    // end finding convolution backward filter workspace size
    // begin allocating gpu array for onvolution backward filter workspace
    CUDA_CALL(hipSetDevice(x6));
    float* x49 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x49, (size_t)(x48 * sizeof(float))));
    // end allocating gpu array for onvolution backward filter workspace
    // begin convolution backward filter pass
    float x50 = 1.0;
    float x51 = 0.0;
    CUDNNCHECK(hipdnnConvolutionBackwardFilter(x7, &x50, x40, x38, x41, x39, x43, x47, x49, x48, &x51, x42, x44));
    // end convolution backward filter pass
    // begin computing ACCUM on GPU for size 18 and type Float at device (pre-rename) x39 with base_operand x93 and addition_operand x235
    CUDA_CALL(hipSetDevice(x6));
    x20<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x11, x44, 18);
    // end computing ACCUM on GPU for size 18 and type Float at device (pre-rename) x39 with base_operand x93 and addition_operand x235
    // begin computing SGD on GPU for size 18 and type Float at device (pre-name) x39 with weight x70, grad x93, and momentum x131
    CUDA_CALL(hipSetDevice(x6));
    x27<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x10, x11, x18, 18);
    // end computing SGD on GPU for size 18 and type Float at device (pre-name) x39 with weight x70, grad x93, and momentum x131
    x19 = x19 + 1;
  }
  if (x6 == 0) {
    // begin copying GPU array x70 to CPU and print for size 18 and type Float
    float* x52 = (float*)malloc(18 * sizeof(float));
    CUDA_CALL(hipMemcpy(x52, x10, (size_t)(18 * sizeof(float)), hipMemcpyDeviceToHost));
    int x53 = 0;
    while (x53 != 18) {
      printf("%f ", x52[x53]);
      x53 = x53 + 1;
    }
    printf("\n");
    // end copying GPU array x70 to CPU and print for size 18 and type Float
  }
  printf("compile");
  CUDNNCHECK(hipdnnDestroy(x7));
  MPICHECK(MPI_Finalize());
  NCCLCHECK(ncclCommDestroy(x4));
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
