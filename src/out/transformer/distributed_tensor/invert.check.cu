#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#include "cudnn_header.h"
#include "nccl_header.h"
#include <string.h>
#include <cblas.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include "mpi_header.h"
/************* Functions **************/
__global__ void x12(float* x13, float x14, int x15) {
  int x16 = gridDim.x * blockDim.x;
  int x17 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x17 < x15) {
    x13[x17] = x14;
    x17 = x17 + x16;
  }
}
__global__ void x20(float* x21, float* x22, int x23) {
  // begin generating kernel function for ACCUM of type Float
  int x24 = gridDim.x * blockDim.x;
  int x25 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x25 < x23) {
    int x26 = x25;
    x21[x26] = x21[x26] + x22[x26];
    x25 = x25 + x24;
  }
  // end generating kernel function for ACCUM of type Float
}
__global__ void x27(float* x28, float* x29, float* x30, int x31) {
  // begin generating kernel function for INVERT_GRAD of type Float
  int x32 = gridDim.x * blockDim.x;
  int x33 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x33 < x31) {
    int x34 = x33;
    float x35 = x28[x34];
    x30[x34] = 0.0 - x29[x34] / (x35 * x35);
    x33 = x33 + x32;
  }
  // end generating kernel function for INVERT_GRAD of type Float
}
__global__ void x36(float* x37, float* x38, float* x39, int x40) {
  // begin generating kernel function for SGD of type Float
  int x41 = gridDim.x * blockDim.x;
  int x42 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x42 < x40) {
    int x43 = x42;
    float x44 = x39[x43] * 0.5 + x38[x43];
    x37[x43] = x37[x43] - x44 * 1.0E-4;
    x39[x43] = x44;
    x42 = x42 + x41;
  }
  // end generating kernel function for SGD of type Float
}
/**************** Snippet ****************/
void Snippet(int x0) {
  // begin setting up the MPI/NCCL environment
  int x1 = 0;
  int x2 = 0;
  MPICHECK(MPI_Init(NULL, NULL));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &x2));
  int x3 = MPI_Comm_size(MPI_COMM_WORLD, &x1);
  MPICHECK(x3);
  MPICHECK(MPI_Barrier(MPI_COMM_WORLD));
  CUDA_CALL(hipSetDevice(x2));
  ncclUniqueId x4;
  NCCLCHECK(ncclGetUniqueId(&x4));
  MPICHECK(MPI_Bcast(&x4, NCCL_UNIQUE_ID_BYTES, MPI_CHAR, 0, MPI_COMM_WORLD));
  ncclComm_t x5;
  NCCLCHECK(ncclCommInitRank(&x5, x1, x4, x2));
  hipStream_t x6;
  CUDA_CALL(hipStreamCreateWithFlags(&x6, hipStreamNonBlocking));
  int x7 = x2;
  // end setting up the MPI/NCCL environment
  // begin initializing random GPU array of size 512 and type Float at device (pre-rename) x39
  float* x8 = (float*)malloc(512 * sizeof(float));
  int x9 = 0;
  while (x9 != 512) {
    x8[x9] = (float)(rand() - RAND_MAX / 2) / (float)RAND_MAX;
    x9 = x9 + 1;
  }
  CUDA_CALL(hipSetDevice(x7));
  float* x10 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x10, (size_t)(512 * sizeof(float))));
  CUDA_CALL(hipMemcpy(x10, x8, (size_t)(512 * sizeof(float)), hipMemcpyHostToDevice));
  // end initializing random GPU array of size 512 and type Float at device (pre-rename) x39
  // begin initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x7));
  float* x11 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x11, (size_t)(512 * sizeof(float))));
  x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x11, 0, 512);
  // end initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  // begin initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x7));
  float* x18 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x18, (size_t)(512 * sizeof(float))));
  x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x18, 0, 512);
  // end initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  int x19 = 0;
  while (x19 != 10) {
    // begin initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
    CUDA_CALL(hipSetDevice(x7));
    float* x45 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x45, (size_t)(512 * sizeof(float))));
    x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x45, 0, 512);
    // end initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
    // begin initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
    CUDA_CALL(hipSetDevice(x7));
    float* x46 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x46, (size_t)(512 * sizeof(float))));
    x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x46, 1, 512);
    // end initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
    // begin computing ACCUM on GPU for size 512 and type Float at device (pre-rename) x39 with base_operand x137 and addition_operand x150
    CUDA_CALL(hipSetDevice(x7));
    x20<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x45, x46, 512);
    // end computing ACCUM on GPU for size 512 and type Float at device (pre-rename) x39 with base_operand x137 and addition_operand x150
    // begin computing INVERT_GRAD on GPU for size 512 and type Float at device (pre-rename) x39 with left_operand x65 and right_operand x137
    CUDA_CALL(hipSetDevice(x7));
    float* x47 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x47, (size_t)(512 * sizeof(float))));
    x27<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x10, x45, x47, 512);
    // end computing INVERT_GRAD on GPU for size 512 and type Float at device (pre-rename) x39 with left_operand x65 and right_operand x137
    // begin computing ACCUM on GPU for size 512 and type Float at device (pre-rename) x39 with base_operand x81 and addition_operand x200
    CUDA_CALL(hipSetDevice(x7));
    x20<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x11, x47, 512);
    // end computing ACCUM on GPU for size 512 and type Float at device (pre-rename) x39 with base_operand x81 and addition_operand x200
    // begin computing SGD on GPU for size 512 and type Float at device (pre-name) x39 with weight x65, grad x81, and momentum x119
    CUDA_CALL(hipSetDevice(x7));
    x36<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x10, x11, x18, 512);
    // end computing SGD on GPU for size 512 and type Float at device (pre-name) x39 with weight x65, grad x81, and momentum x119
    x19 = x19 + 1;
  }
  // Only declare recv buffer if this is the root
  bool x48 = x7 == 0;
  float* x49 = x48 ? ({
    float* x50 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x50, (size_t)(1024 * sizeof(float))));
    x50;
  }) : ({
    float* x51 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x51, (size_t)0));
    x51;
  });
  // Gather by groups of NCCL send/recv
  NCCLCHECK(ncclGroupStart());
  ncclResult_t x52 = ncclSend(x10, (size_t)1024, ncclFloat32, 0, x5, x6);
  NCCLCHECK(x52);
  if (x48) {
    int x53 = x1;
    int x54 = 0;
    while (x54 != x53) {
      int x55 = x54;
      NCCLCHECK(ncclRecv(x49 + x55 * 512, (size_t)1024, ncclFloat32, x55, x5, x6));
      x54 = x54 + 1;
    }
  }
  NCCLCHECK(ncclGroupEnd());
  // print the array only if this is the root
  if (x48) {
    // begin copying GPU array x312 to CPU and print for size 1024 and type Float
    float* x56 = (float*)malloc(1024 * sizeof(float));
    CUDA_CALL(hipMemcpy(x56, x49, (size_t)(1024 * sizeof(float)), hipMemcpyDeviceToHost));
    int x57 = 0;
    while (x57 != 1024) {
      printf("%f ", x56[x57]);
      x57 = x57 + 1;
    }
    printf("\n");
    // end copying GPU array x312 to CPU and print for size 1024 and type Float
  }
  printf("compile\n");
  MPICHECK(MPI_Finalize());
  NCCLCHECK(ncclCommDestroy(x5));
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
