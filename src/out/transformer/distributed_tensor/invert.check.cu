#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#include "nccl_header.h"
#include <string.h>
#include <cblas.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include "mpi_header.h"
/************* Functions **************/
__global__ void x12(float* x13, float x14, int x15) {
  int x16 = gridDim.x * blockDim.x;
  int x17 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x17 < x15) {
    x13[x17] = x14;
    x17 = x17 + x16;
  }
}
__global__ void x20(float* x21, float* x22, float* x23, int x24) {
  // begin generating kernel function for MULT of type Float
  int x25 = gridDim.x * blockDim.x;
  int x26 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x26 < x24) {
    int x27 = x26;
    x23[x27] = x21[x27] * x22[x27];
    x26 = x26 + x25;
  }
  // end generating kernel function for MULT of type Float
}
__global__ void x28(float* x29, float* x30, float* x31, int x32) {
  // begin generating kernel function for DIV of type Float
  int x33 = gridDim.x * blockDim.x;
  int x34 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x34 < x32) {
    int x35 = x34;
    x31[x35] = x29[x35] / x30[x35];
    x34 = x34 + x33;
  }
  // end generating kernel function for DIV of type Float
}
__global__ void x36(float* x37, float* x38, int x39) {
  // begin generating kernel function for ACCUM of type Float
  int x40 = gridDim.x * blockDim.x;
  int x41 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x41 < x39) {
    int x42 = x41;
    x37[x42] = x37[x42] + x38[x42];
    x41 = x41 + x40;
  }
  // end generating kernel function for ACCUM of type Float
}
__global__ void x43(float* x44, float* x45, float* x46, int x47) {
  // begin generating kernel function for SGD of type Float
  int x48 = gridDim.x * blockDim.x;
  int x49 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x49 < x47) {
    int x50 = x49;
    float x51 = x46[x50] * 0.5 + x45[x50];
    x44[x50] = x44[x50] - x51 * 1.0E-4;
    x46[x50] = x51;
    x49 = x49 + x48;
  }
  // end generating kernel function for SGD of type Float
}
/**************** Snippet ****************/
void Snippet(int x0) {
  // begin setting up the MPI/NCCL environment
  int x1 = 0;
  int x2 = 0;
  MPICHECK(MPI_Init(NULL, NULL));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &x2));
  int x3 = MPI_Comm_size(MPI_COMM_WORLD, &x1);
  MPICHECK(x3);
  MPICHECK(MPI_Barrier(MPI_COMM_WORLD));
  CUDA_CALL(hipSetDevice(x2));
  ncclUniqueId x4;
  NCCLCHECK(ncclGetUniqueId(&x4));
  MPICHECK(MPI_Bcast(&x4, NCCL_UNIQUE_ID_BYTES, MPI_CHAR, 0, MPI_COMM_WORLD));
  ncclComm_t x5;
  NCCLCHECK(ncclCommInitRank(&x5, x1, x4, x2));
  hipStream_t x6;
  CUDA_CALL(hipStreamCreateWithFlags(&x6, hipStreamNonBlocking));
  int x7 = x2;
  // end setting up the MPI/NCCL environment
  // begin initializing random GPU array of size 512 and type Float at device (pre-rename) x39
  float* x8 = (float*)malloc(512 * sizeof(float));
  int x9 = 0;
  while (x9 != 512) {
    x8[x9] = (float)(rand() - RAND_MAX / 2) / (float)RAND_MAX;
    x9 = x9 + 1;
  }
  CUDA_CALL(hipSetDevice(x7));
  float* x10 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x10, (size_t)(512 * sizeof(float))));
  CUDA_CALL(hipMemcpy(x10, x8, (size_t)(512 * sizeof(float)), hipMemcpyHostToDevice));
  // end initializing random GPU array of size 512 and type Float at device (pre-rename) x39
  // begin initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x7));
  float* x11 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x11, (size_t)(512 * sizeof(float))));
  x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x11, 0, 512);
  // end initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  // begin initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x7));
  float* x18 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x18, (size_t)(512 * sizeof(float))));
  x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x18, 0, 512);
  // end initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  int x19 = 0;
  while (x19 != 10) {
    // begin computing MULT on GPU for size 512 and type Float at device (pre-rename) x39 with left_operand x65 and right_operand x65
    CUDA_CALL(hipSetDevice(x7));
    float* x52 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x52, (size_t)(512 * sizeof(float))));
    x20<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x10, x10, x52, 512);
    // end computing MULT on GPU for size 512 and type Float at device (pre-rename) x39 with left_operand x65 and right_operand x65
    // begin computing DIV on GPU for size 512 and type Float at device (pre-rename) x39 with left_operand x81 and right_operand x137
    CUDA_CALL(hipSetDevice(x7));
    float* x53 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x53, (size_t)(512 * sizeof(float))));
    x28<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x11, x52, x53, 512);
    // end computing DIV on GPU for size 512 and type Float at device (pre-rename) x39 with left_operand x81 and right_operand x137
    // begin computing ACCUM on GPU for size 512 and type Float at device (pre-rename) x39 with base_operand x81 and addition_operand x181
    CUDA_CALL(hipSetDevice(x7));
    x36<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x11, x53, 512);
    // end computing ACCUM on GPU for size 512 and type Float at device (pre-rename) x39 with base_operand x81 and addition_operand x181
    // begin computing SGD on GPU for size 512 and type Float at device (pre-name) x39 with weight x65, grad x81, and momentum x119
    CUDA_CALL(hipSetDevice(x7));
    x43<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x10, x11, x18, 512);
    // end computing SGD on GPU for size 512 and type Float at device (pre-name) x39 with weight x65, grad x81, and momentum x119
    x19 = x19 + 1;
  }
  // Only declare recv buffer if this is the root
  bool x54 = x7 == 0;
  float* x55 = x54 ? ({
    float* x56 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x56, (size_t)(1024 * sizeof(float))));
    x56;
  }) : ({
    float* x57 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x57, (size_t)0));
    x57;
  });
  // Gather by groups of NCCL send/recv
  NCCLCHECK(ncclGroupStart());
  ncclResult_t x58 = ncclSend(x10, (size_t)1024, ncclFloat32, 0, x5, x6);
  NCCLCHECK(x58);
  if (x54) {
    int x59 = x1;
    int x60 = 0;
    while (x60 != x59) {
      int x61 = x60;
      NCCLCHECK(ncclRecv(x55 + x61 * 512, (size_t)1024, ncclFloat32, x61, x5, x6));
      x60 = x60 + 1;
    }
  }
  NCCLCHECK(ncclGroupEnd());
  // print the array only if this is the root
  if (x54) {
    // begin copying GPU array x321 to CPU and print for size 1024 and type Float
    float* x62 = (float*)malloc(1024 * sizeof(float));
    CUDA_CALL(hipMemcpy(x62, x55, (size_t)(1024 * sizeof(float)), hipMemcpyDeviceToHost));
    int x63 = 0;
    while (x63 != 1024) {
      printf("%f ", x62[x63]);
      x63 = x63 + 1;
    }
    printf("\n");
    // end copying GPU array x321 to CPU and print for size 1024 and type Float
  }
  printf("compile\n");
  MPICHECK(MPI_Finalize());
  NCCLCHECK(ncclCommDestroy(x5));
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
