#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#include "cudnn_header.h"
#include "nccl_header.h"
#include <string.h>
#include <cblas.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include "mpi_header.h"
/************* Functions **************/
__global__ void x12(float* x13, float x14, int x15) {
  int x16 = gridDim.x * blockDim.x;
  int x17 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x17 < x15) {
    x13[x17] = x14;
    x17 = x17 + x16;
  }
}
__global__ void x20(float* x21, float* x22, float* x23, int x24) {
  // begin generating kernel function for MULT of type Float
  int x25 = gridDim.x * blockDim.x;
  int x26 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x26 < x24) {
    int x27 = x26;
    x23[x27] = x21[x27] * x22[x27];
    x26 = x26 + x25;
  }
  // end generating kernel function for MULT of type Float
}
__global__ void x28(float* x29, float* x30, int x31) {
  // begin generating kernel function for ACCUM of type Float
  int x32 = gridDim.x * blockDim.x;
  int x33 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x33 < x31) {
    int x34 = x33;
    x29[x34] = x29[x34] + x30[x34];
    x33 = x33 + x32;
  }
  // end generating kernel function for ACCUM of type Float
}
__global__ void x35(float* x36, float* x37, float* x38, int x39) {
  // begin generating kernel function for SGD of type Float
  int x40 = gridDim.x * blockDim.x;
  int x41 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x41 < x39) {
    int x42 = x41;
    float x43 = x38[x42] * 0.5 + x37[x42];
    x36[x42] = x36[x42] - x43 * 1.0E-4;
    x38[x42] = x43;
    x41 = x41 + x40;
  }
  // end generating kernel function for SGD of type Float
}
/**************** Snippet ****************/
void Snippet(int x0) {
  // begin setting up the MPI/NCCL environment
  int x1 = 0;
  int x2 = 0;
  MPICHECK(MPI_Init(NULL, NULL));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &x2));
  int x3 = MPI_Comm_size(MPI_COMM_WORLD, &x1);
  MPICHECK(x3);
  MPICHECK(MPI_Barrier(MPI_COMM_WORLD));
  CUDA_CALL(hipSetDevice(x2));
  ncclUniqueId x4;
  NCCLCHECK(ncclGetUniqueId(&x4));
  MPICHECK(MPI_Bcast(&x4, NCCL_UNIQUE_ID_BYTES, MPI_CHAR, 0, MPI_COMM_WORLD));
  ncclComm_t x5;
  NCCLCHECK(ncclCommInitRank(&x5, x1, x4, x2));
  hipStream_t x6;
  CUDA_CALL(hipStreamCreateWithFlags(&x6, hipStreamNonBlocking));
  int x7 = x2;
  // end setting up the MPI/NCCL environment
  // begin initializing random GPU array of size 512 and type Float at device (pre-rename) x39
  float* x8 = (float*)malloc(512 * sizeof(float));
  int x9 = 0;
  while (x9 != 512) {
    x8[x9] = (float)(rand() - RAND_MAX / 2) / (float)RAND_MAX;
    x9 = x9 + 1;
  }
  CUDA_CALL(hipSetDevice(x7));
  float* x10 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x10, (size_t)(512 * sizeof(float))));
  CUDA_CALL(hipMemcpy(x10, x8, (size_t)(512 * sizeof(float)), hipMemcpyHostToDevice));
  // end initializing random GPU array of size 512 and type Float at device (pre-rename) x39
  // begin initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x7));
  float* x11 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x11, (size_t)(512 * sizeof(float))));
  x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x11, 0, 512);
  // end initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  // begin initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x7));
  float* x18 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x18, (size_t)(512 * sizeof(float))));
  x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x18, 0, 512);
  // end initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  int x19 = 0;
  while (x19 != 10) {
    // begin initializing random GPU array of size 512 and type Float at device (pre-rename) x39
    float* x44 = (float*)malloc(512 * sizeof(float));
    int x45 = 0;
    while (x45 != 512) {
      x44[x45] = (float)(rand() - RAND_MAX / 2) / (float)RAND_MAX;
      x45 = x45 + 1;
    }
    CUDA_CALL(hipSetDevice(x7));
    float* x46 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x46, (size_t)(512 * sizeof(float))));
    CUDA_CALL(hipMemcpy(x46, x44, (size_t)(512 * sizeof(float)), hipMemcpyHostToDevice));
    // end initializing random GPU array of size 512 and type Float at device (pre-rename) x39
    // begin initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
    CUDA_CALL(hipSetDevice(x7));
    float* x47 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x47, (size_t)(512 * sizeof(float))));
    x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x47, 1, 512);
    // end initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
    // begin computing MULT on GPU for size 512 and type Float at device (pre-rename) x39 with left_operand x157 and right_operand x173
    CUDA_CALL(hipSetDevice(x7));
    float* x48 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x48, (size_t)(512 * sizeof(float))));
    x20<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x46, x47, x48, 512);
    // end computing MULT on GPU for size 512 and type Float at device (pre-rename) x39 with left_operand x157 and right_operand x173
    // begin computing ACCUM on GPU for size 512 and type Float at device (pre-rename) x39 with base_operand x81 and addition_operand x186
    CUDA_CALL(hipSetDevice(x7));
    x28<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x11, x48, 512);
    // end computing ACCUM on GPU for size 512 and type Float at device (pre-rename) x39 with base_operand x81 and addition_operand x186
    // begin computing SGD on GPU for size 512 and type Float at device (pre-name) x39 with weight x65, grad x81, and momentum x119
    CUDA_CALL(hipSetDevice(x7));
    x35<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x10, x11, x18, 512);
    // end computing SGD on GPU for size 512 and type Float at device (pre-name) x39 with weight x65, grad x81, and momentum x119
    x19 = x19 + 1;
  }
  // Only declare recv buffer if this is the root
  bool x49 = x7 == 0;
  float* x50 = x49 ? ({
    float* x51 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x51, (size_t)(1024 * sizeof(float))));
    x51;
  }) : ({
    float* x52 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x52, (size_t)0));
    x52;
  });
  // Gather by groups of NCCL send/recv
  NCCLCHECK(ncclGroupStart());
  ncclResult_t x53 = ncclSend(x10, (size_t)1024, ncclFloat32, 0, x5, x6);
  NCCLCHECK(x53);
  if (x49) {
    int x54 = x1;
    int x55 = 0;
    while (x55 != x54) {
      int x56 = x55;
      NCCLCHECK(ncclRecv(x50 + x56 * 512, (size_t)1024, ncclFloat32, x56, x5, x6));
      x55 = x55 + 1;
    }
  }
  NCCLCHECK(ncclGroupEnd());
  // print the array only if this is the root
  if (x49) {
    // begin copying GPU array x326 to CPU and print for size 1024 and type Float
    float* x57 = (float*)malloc(1024 * sizeof(float));
    CUDA_CALL(hipMemcpy(x57, x50, (size_t)(1024 * sizeof(float)), hipMemcpyDeviceToHost));
    int x58 = 0;
    while (x58 != 1024) {
      printf("%f ", x57[x58]);
      x58 = x58 + 1;
    }
    printf("\n");
    // end copying GPU array x326 to CPU and print for size 1024 and type Float
  }
  printf("compile\n");
  MPICHECK(MPI_Finalize());
  NCCLCHECK(ncclCommDestroy(x5));
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
