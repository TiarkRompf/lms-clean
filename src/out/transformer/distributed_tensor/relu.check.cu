#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#include "cudnn_header.h"
#include "nccl_header.h"
#include <string.h>
#include <cblas.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include "mpi_header.h"
/************* Functions **************/
__global__ void x12(float* x13, float x14, int x15) {
  int x16 = gridDim.x * blockDim.x;
  int x17 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x17 < x15) {
    x13[x17] = x14;
    x17 = x17 + x16;
  }
}
__global__ void x20(float* x21, float* x22, int x23) {
  // begin generating kernel function for ACCUM of type Float
  int x24 = gridDim.x * blockDim.x;
  int x25 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x25 < x23) {
    int x26 = x25;
    x21[x26] = x21[x26] + x22[x26];
    x25 = x25 + x24;
  }
  // end generating kernel function for ACCUM of type Float
}
__global__ void x27(float* x28, float* x29, float* x30, int x31) {
  // begin generating kernel function for RELU_GRAD of type Float
  int x32 = gridDim.x * blockDim.x;
  int x33 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x33 < x31) {
    int x34 = x33;
    x30[x34] = x29[x34] > 0.0 ? x28[x34] : 0.0;
    x33 = x33 + x32;
  }
  // end generating kernel function for RELU_GRAD of type Float
}
__global__ void x35(float* x36, float* x37, float* x38, int x39) {
  // begin generating kernel function for SGD of type Float
  int x40 = gridDim.x * blockDim.x;
  int x41 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x41 < x39) {
    int x42 = x41;
    float x43 = x38[x42] * 0.5 + x37[x42];
    x36[x42] = x36[x42] - x43 * 1.0E-4;
    x38[x42] = x43;
    x41 = x41 + x40;
  }
  // end generating kernel function for SGD of type Float
}
/**************** Snippet ****************/
void Snippet(int x0) {
  // begin setting up the MPI/NCCL environment
  int x1 = 0;
  int x2 = 0;
  MPICHECK(MPI_Init(NULL, NULL));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &x2));
  int x3 = MPI_Comm_size(MPI_COMM_WORLD, &x1);
  MPICHECK(x3);
  MPICHECK(MPI_Barrier(MPI_COMM_WORLD));
  CUDA_CALL(hipSetDevice(x2));
  ncclUniqueId x4;
  NCCLCHECK(ncclGetUniqueId(&x4));
  MPICHECK(MPI_Bcast(&x4, NCCL_UNIQUE_ID_BYTES, MPI_CHAR, 0, MPI_COMM_WORLD));
  ncclComm_t x5;
  NCCLCHECK(ncclCommInitRank(&x5, x1, x4, x2));
  hipStream_t x6;
  CUDA_CALL(hipStreamCreateWithFlags(&x6, hipStreamNonBlocking));
  int x7 = x2;
  // end setting up the MPI/NCCL environment
  // begin initializing random GPU array of size 512 and type Float at device (pre-rename) x39
  float* x8 = (float*)malloc(512 * sizeof(float));
  int x9 = 0;
  while (x9 != 512) {
    x8[x9] = (float)(rand() - RAND_MAX / 2) / (float)RAND_MAX;
    x9 = x9 + 1;
  }
  CUDA_CALL(hipSetDevice(x7));
  float* x10 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x10, (size_t)(512 * sizeof(float))));
  CUDA_CALL(hipMemcpy(x10, x8, (size_t)(512 * sizeof(float)), hipMemcpyHostToDevice));
  // end initializing random GPU array of size 512 and type Float at device (pre-rename) x39
  // begin initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x7));
  float* x11 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x11, (size_t)(512 * sizeof(float))));
  x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x11, 0, 512);
  // end initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  // begin initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x7));
  float* x18 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x18, (size_t)(512 * sizeof(float))));
  x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x18, 0, 512);
  // end initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  int x19 = 0;
  while (x19 != 10) {
    // begin initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
    CUDA_CALL(hipSetDevice(x7));
    float* x44 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x44, (size_t)(512 * sizeof(float))));
    x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x44, 0, 512);
    // end initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
    // begin initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
    CUDA_CALL(hipSetDevice(x7));
    float* x45 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x45, (size_t)(512 * sizeof(float))));
    x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x45, 1, 512);
    // end initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
    // begin computing ACCUM on GPU for size 512 and type Float at device (pre-rename) x39 with base_operand x137 and addition_operand x150
    CUDA_CALL(hipSetDevice(x7));
    x20<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x44, x45, 512);
    // end computing ACCUM on GPU for size 512 and type Float at device (pre-rename) x39 with base_operand x137 and addition_operand x150
    // begin computing RELU_GRAD on GPU for size 512 and type Float at device (pre-rename) x39 with left_operand x137 and right_operand x65
    CUDA_CALL(hipSetDevice(x7));
    float* x46 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x46, (size_t)(512 * sizeof(float))));
    x27<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x44, x10, x46, 512);
    // end computing RELU_GRAD on GPU for size 512 and type Float at device (pre-rename) x39 with left_operand x137 and right_operand x65
    // begin computing ACCUM on GPU for size 512 and type Float at device (pre-rename) x39 with base_operand x81 and addition_operand x200
    CUDA_CALL(hipSetDevice(x7));
    x20<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x11, x46, 512);
    // end computing ACCUM on GPU for size 512 and type Float at device (pre-rename) x39 with base_operand x81 and addition_operand x200
    // begin computing SGD on GPU for size 512 and type Float at device (pre-name) x39 with weight x65, grad x81, and momentum x119
    CUDA_CALL(hipSetDevice(x7));
    x35<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x10, x11, x18, 512);
    // end computing SGD on GPU for size 512 and type Float at device (pre-name) x39 with weight x65, grad x81, and momentum x119
    x19 = x19 + 1;
  }
  // Only declare recv buffer if this is the root
  bool x47 = x7 == 0;
  float* x48 = x47 ? ({
    float* x49 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x49, (size_t)(1024 * sizeof(float))));
    x49;
  }) : ({
    float* x50 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x50, (size_t)0));
    x50;
  });
  // Gather by groups of NCCL send/recv
  NCCLCHECK(ncclGroupStart());
  ncclResult_t x51 = ncclSend(x10, (size_t)1024, ncclFloat32, 0, x5, x6);
  NCCLCHECK(x51);
  if (x47) {
    int x52 = x1;
    int x53 = 0;
    while (x53 != x52) {
      int x54 = x53;
      NCCLCHECK(ncclRecv(x48 + x54 * 512, (size_t)1024, ncclFloat32, x54, x5, x6));
      x53 = x53 + 1;
    }
  }
  NCCLCHECK(ncclGroupEnd());
  // print the array only if this is the root
  if (x47) {
    // begin copying GPU array x313 to CPU and print for size 1024 and type Float
    float* x55 = (float*)malloc(1024 * sizeof(float));
    CUDA_CALL(hipMemcpy(x55, x48, (size_t)(1024 * sizeof(float)), hipMemcpyDeviceToHost));
    int x56 = 0;
    while (x56 != 1024) {
      printf("%f ", x55[x56]);
      x56 = x56 + 1;
    }
    printf("\n");
    // end copying GPU array x313 to CPU and print for size 1024 and type Float
  }
  printf("compile\n");
  MPICHECK(MPI_Finalize());
  NCCLCHECK(ncclCommDestroy(x5));
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
