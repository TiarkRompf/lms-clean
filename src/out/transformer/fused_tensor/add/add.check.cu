#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#include <string.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
/************* Functions **************/
__global__ void x4(int x5, int x6, int x7) {
  int x8 = gridDim.x * blockDim.x;
  int x9 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x9 < x7) {
    x6[x9] = x2[x9];
    x9 = x9 + x8;
  }
}
__global__ void x11(int x12, int x13, int x14) {
  int x15 = gridDim.x * blockDim.x;
  int x16 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x16 < x14) {
    x13[x16] = x2[x16] - 1;
    x16 = x16 + x15;
  }
}
/**************** Snippet ****************/
void Snippet(int x0) {
  int* x1 = (int*)malloc(10 * sizeof(int));
  scan_int_array(x1, 10, "input");
  CUDA_CALL(hipSetDevice(0));
  int* x2 = (int*)malloc(0 * sizeof(int));
  CUDA_CALL(hipMalloc(&x2, (size_t)(10 * sizeof(int))));
  CUDA_CALL(hipMemcpy(x2, x1, (size_t)(10 * sizeof(int)), hipMemcpyHostToDevice));
  int* x3 = (int*)malloc(0 * sizeof(int));
  CUDA_CALL(hipMalloc(&x3, (size_t)(10 * sizeof(int))));
  int* x10 = (int*)malloc(0 * sizeof(int));
  CUDA_CALL(hipMalloc(&x10, (size_t)(10 * sizeof(int))));
  show_tensor(x4<<<dim3(0, 1, 1), dim3(0, 1, 1)>>>(x1, x3, 10));
  show_tensor(x11<<<dim3(0, 1, 1), dim3(0, 1, 1)>>>(x1, x10, 10));
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
