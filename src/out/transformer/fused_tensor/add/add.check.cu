#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#include <string.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
/************* Functions **************/
__global__ void x2(int x3, int x4, int x5) {
  int x6 = gridDim.x * blockDim.x;
  int x7 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x7 < x5) {
    x3[x7] = x3[x7];
    x7 = x7 + x6;
  }
}
__global__ void x8(int x9, int x10, int x11) {
  int x12 = gridDim.x * blockDim.x;
  int x13 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x13 < x11) {
    x9[x13] = x9[x13] - 1;
    x13 = x13 + x12;
  }
}
/**************** Snippet ****************/
void Snippet(int x0) {
  int* x1 = (int*)malloc(0 * sizeof(int));
  CUDA_CALL(hipMalloc(&x1, (size_t)(10 * sizeof(int))));
  show_tensor(x2<<<dim3(0, 1, 1), dim3(0, 1, 1)>>>(x1, 0, 10));
  show_tensor(x8<<<dim3(0, 1, 1), dim3(0, 1, 1)>>>(x1, 0, 10));
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
