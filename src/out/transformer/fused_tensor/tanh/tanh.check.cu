#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#include <string.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include <math.h>
/************* Functions **************/
__global__ void x2(int x3, int x4, int x5) {
  int x6 = gridDim.x * blockDim.x;
  int x7 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x7 < x5) {
    x3[x7] = tanh(x3[x7]);
    x7 = x7 + x6;
  }
}
/**************** Snippet ****************/
void Snippet(int x0) {
  int* x1 = (int*)malloc(0 * sizeof(int));
  CUDA_CALL(hipMalloc(&x1, (size_t)(10 * sizeof(int))));
  show_tensor(x2<<<dim3(0, 1, 1), dim3(0, 1, 1)>>>(x1, 0, 10));
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
