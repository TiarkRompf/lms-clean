#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include <math.h>
/************* Functions **************/
__global__ void x3(int x4, int x5, int x6) {
  int x7 = gridDim.x * blockDim.x;
  int x8 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x8 < x6) {
    x4[x8] = x1;
    x8 = x8 + x7;
  }
}
/**************** Snippet ****************/
void Snippet(int x0) {
  int x1 = tanh(0);
  int* x2 = (int*)malloc(0 * sizeof(int));
  CUDA_CALL(hipMalloc(&x2, (lms.thirdparty.size_ttypeless$sizet)(10 * sizeof(Int))));
  show_tensor(x3(x2, 0, 0, dim3(0, 1, 1), dim3(0, 1, 1)));
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
