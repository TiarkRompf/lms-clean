#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
/************* Functions **************/
__global__ void x2(int x3, int x4, int x5) {
  int x6 = gridDim.x * blockDim.x;
  int x7 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x7 < x5) {
    int x8 = x7;
    x3[x7] = x8 < 5 ? x3[x8] + 1 : x3[x8];
    x7 = x7 + x6;
  }
}
/**************** Snippet ****************/
void Snippet(int x0) {
  int* x1 = (int*)malloc(0 * sizeof(int));
  CUDA_CALL(hipMalloc(&x1, (lms.thirdparty.size_ttypeless$sizet)(10 * sizeof(Int))));
  show_tensor(x2(x1, 0, 20, dim3(0, 1, 1), dim3(0, 1, 1)));
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
