/*****************************************
Emitting C Generated Code
*******************************************/
#include "nccl_header.h"
#include <string.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include "mpi_header.h"
/**************** Snippet ****************/
void Snippet(int x0) {
  int x1 = 0;
  int x2 = 0;
  MPICHECK(MPI_Init(NULL, NULL));
  int x3 = MPI_Comm_rank(MPI_COMM_WORLD, &x1);
  MPICHECK(x3);
  int x4 = MPI_Comm_size(MPI_COMM_WORLD, &x2);
  MPICHECK(x4);
  printf("myRank: %d, nRanks: %d\n", x1, x2);
  ncclUniqueId x5;
  ncclComm_t x6;
  hipStream_t x7;
  if (x1 == 0) NCCLCHECK(ncclGetUniqueId(&x5));
  MPICHECK(MPI_Bcast(&x5, NCCL_UNIQUE_ID_BYTES, MPI_BYTE, 0, MPI_COMM_WORLD));
  float* x8 = (float*)malloc(1024 * sizeof(float));
  int x9 = 0;
  while (x9 != 1024) {
    x8[x9] = 2.0;
    x9 = x9 + 1;
  }
  CUDA_CALL(hipSetDevice(x1));
  float* x10 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x10, (size_t)(1024 * sizeof(float))));
  CUDA_CALL(hipMemcpy(x10, x8, (size_t)(1024 * sizeof(float)), hipMemcpyHostToDevice));
  float** x11 = (float**)malloc(x2 * sizeof(float*));
  int x12 = x2;
  int x13 = 0;
  while (x13 != x12) {
    CUDA_CALL(hipMalloc(&x11[x13], (size_t)(1024 * sizeof(float))));
    x13 = x13 + 1;
  }
  hipError_t x14 = hipStreamCreateWithFlags(&x7, hipStreamDefault);
  CUDA_CALL(x14);
  ncclResult_t x15 = ncclCommInitRank(&x6, x2, x5, x1);
  NCCLCHECK(x15);
  NCCLCHECK(ncclGroupStart());
  if (x1 == 0) {
    int x16 = x2;
    int x17 = 0;
    while (x17 != x16) {
      int x18 = x17;
      NCCLCHECK(ncclRecv(x11[x18], 1024, ncclFloat, x18, x6, x7));
      x17 = x17 + 1;
    }
  }
  NCCLCHECK(ncclSend(x10, 1024, ncclFloat, 0, x6, x7));
  NCCLCHECK(ncclGroupEnd());
  CUDA_CALL(hipStreamSynchronize(x7));
  int x19 = 0;
  if (x1 == 0) {
    int x20 = x2;
    int x21 = 0;
    while (x21 != x20) {
      CUDA_CALL(hipMemcpy(x8, x11[x21], (size_t)(1024 * sizeof(float)), hipMemcpyDeviceToHost));
      int x22 = 0;
      while (x22 != 1024) {
        if (x8[x22] != 2) x19 = x19 + 1;
        x22 = x22 + 1;
      }
      x21 = x21 + 1;
    }
  }
  CUDA_CALL(hipFree(x10));
  int x23 = x2;
  int x24 = 0;
  while (x24 != x23) {
    CUDA_CALL(hipFree(x11[x24]));
    x24 = x24 + 1;
  }
  NCCLCHECK(ncclCommDestroy(x6));
  MPICHECK(MPI_Finalize());
  if (x19 != 0) printf("[MPI Rank %d] Found %d errors.\n", x1, x19);
  else printf("[MPI Rank %d] Success \n", x1);
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
