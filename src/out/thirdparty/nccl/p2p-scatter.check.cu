/*****************************************
Emitting C Generated Code
*******************************************/
#include "nccl_header.h"
#include <string.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include "mpi_header.h"
/**************** Snippet ****************/
void Snippet(int x0) {
  int x1 = 0;
  int x2 = 0;
  MPICHECK(MPI_Init(NULL, NULL));
  int x3 = MPI_Comm_rank(MPI_COMM_WORLD, &x1);
  MPICHECK(x3);
  int x4 = MPI_Comm_size(MPI_COMM_WORLD, &x2);
  MPICHECK(x4);
  printf("myRank: %d, nRanks: %d\n", x1, x2);
  ncclUniqueId x5;
  ncclComm_t x6;
  hipStream_t x7;
  if (x1 == 0) NCCLCHECK(ncclGetUniqueId(&x5));
  MPICHECK(MPI_Bcast(&x5, NCCL_UNIQUE_ID_BYTES, MPI_BYTE, 0, MPI_COMM_WORLD));
  float* x8 = (float*)malloc(2014 * sizeof(float));
  int x9 = 0;
  while (x9 != 2014) {
    x8[x9] = 2.0;
    x9 = x9 + 1;
  }
  CUDA_CALL(hipSetDevice(x1));
  float** x10 = (float**)malloc(x2 * sizeof(float*));
  int x11 = x2;
  int x12 = 0;
  while (x12 != x11) {
    int x13 = x12;
    CUDA_CALL(hipMalloc(&x10[x13], 2014));
    CUDA_CALL(hipMemcpy(x10[x13], x8, (size_t)(2014 * sizeof(float)), hipMemcpyHostToDevice));
    x12 = x12 + 1;
  }
  float* x14 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x14, (size_t)(2014 * sizeof(float))));
  hipError_t x15 = hipStreamCreateWithFlags(&x7, hipStreamDefault);
  CUDA_CALL(x15);
  ncclResult_t x16 = ncclCommInitRank(&x6, x2, x5, x1);
  NCCLCHECK(x16);
  NCCLCHECK(ncclGroupStart());
  if (x1 == 0) {
    int x17 = x2;
    int x18 = 0;
    while (x18 != x17) {
      int x19 = x18;
      NCCLCHECK(ncclSend(x10[x19], 2014, ncclFloat, x19, x6, x7));
      x18 = x18 + 1;
    }
  }
  NCCLCHECK(ncclRecv(x14, 2014, ncclFloat, 0, x6, x7));
  NCCLCHECK(ncclGroupEnd());
  CUDA_CALL(hipStreamSynchronize(x7));
  CUDA_CALL(hipMemcpy(x8, x14, (size_t)(2014 * sizeof(float)), hipMemcpyDeviceToHost));
  int x20 = 0;
  int x21 = 0;
  while (x21 != 2014) {
    if (x8[0] != 2) x20 = x20 + 1;
    x21 = x21 + 1;
  }
  int x22 = x2;
  int x23 = 0;
  while (x23 != x22) {
    CUDA_CALL(hipFree(x10[x23]));
    x23 = x23 + 1;
  }
  CUDA_CALL(hipFree(x14));
  NCCLCHECK(ncclCommDestroy(x6));
  MPICHECK(MPI_Finalize());
  if (x20 != 0) printf("[MPI Rank %d] Found %d errors.\n", x1, x20);
  else printf("[MPI Rank %d] Success \n", x1);
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
