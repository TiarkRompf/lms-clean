/*****************************************
Emitting C Generated Code
*******************************************/
#include "nccl_header.h"
#include <string.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include "mpi_header.h"
/**************** Snippet ****************/
void Snippet(int x0) {
  ncclComm_t x1;
  int x2[1] = { 0 };
  hipStream_t x3;
  CUDA_CALL(hipSetDevice(0));
  float* x4 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x4, (size_t)(33554432 * sizeof(float))));
  float* x5 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x5, (size_t)(33554432 * sizeof(float))));
  CUDA_CALL(hipMemset(x4, 1, (size_t)(33554432 * sizeof(float))));
  CUDA_CALL(hipMemset(x5, 0, (size_t)(33554432 * sizeof(float))));
  CUDA_CALL(hipStreamCreate(&x3));
  NCCLCHECK(ncclCommInitAll(&x1, 1, x2));
  NCCLCHECK(ncclAllReduce(x4, x5, 33554432, ncclFloat, ncclSum, x1, x3));
  CUDA_CALL(hipSetDevice(0));
  CUDA_CALL(hipStreamSynchronize(x3));
  CUDA_CALL(hipFree(x4));
  CUDA_CALL(hipFree(x5));
  NCCLCHECK(ncclCommDestroy(x1));
  printf("Success \n");
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
