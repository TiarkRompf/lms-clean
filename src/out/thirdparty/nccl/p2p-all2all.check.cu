/*****************************************
Emitting C Generated Code
*******************************************/
#include "nccl_header.h"
#include <string.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include "mpi_header.h"
/**************** Snippet ****************/
void Snippet(int x0) {
  int x1 = 0;
  int x2 = 0;
  MPICHECK(MPI_Init(NULL, NULL));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &x1));
  int x3 = MPI_Comm_size(MPI_COMM_WORLD, &x2);
  MPICHECK(x3);
  ncclUniqueId x4;
  if (x1 == 0) NCCLCHECK(ncclGetUniqueId(&x4));
  MPICHECK(MPI_Bcast(&x4, NCCL_UNIQUE_ID_BYTES, MPI_BYTE, 0, MPI_COMM_WORLD));
  CUDA_CALL(hipSetDevice(0));
  hipStream_t x5;
  hipError_t x6 = hipStreamCreateWithFlags(&x5, hipStreamDefault);
  CUDA_CALL(x6);
  ncclComm_t x7;
  ncclResult_t x8 = ncclCommInitRank(&x7, x2, x4, x1);
  NCCLCHECK(x8);
  float* x9 = (float*)malloc(33554432 * sizeof(float));
  int x10 = 0;
  while (x10 != 33554432) {
    x9[x10] = 1.0;
    x10 = x10 + 1;
  }
  float** x11 = (float**)malloc(x2 * sizeof(float*));
  int x12 = x2;
  int x13 = 0;
  while (x13 != x12) {
    int x14 = x13;
    CUDA_CALL(hipMalloc(&x11[x14], 33554432));
    CUDA_CALL(hipMemcpy(x11[x14], x9, 33554432, hipMemcpyHostToDevice));
    x13 = x13 + 1;
  }
  float** x15 = (float**)malloc(x2 * sizeof(float*));
  int x16 = x2;
  int x17 = 0;
  while (x17 != x16) {
    CUDA_CALL(hipMalloc(&x15[x17], 33554432));
    x17 = x17 + 1;
  }
  NCCLCHECK(ncclGroupStart());
  if (x1 == 0) {
    int x18 = x2;
    int x19 = 0;
    while (x19 != x18) {
      int x20 = x19;
      NCCLCHECK(ncclSend(x11[x20], 33554432, ncclFloat, x20, x7, x5));
      NCCLCHECK(ncclRecv(x15[x20], 33554432, ncclFloat, x20, x7, x5));
      x19 = x19 + 1;
    }
  }
  NCCLCHECK(ncclGroupEnd());
  CUDA_CALL(hipStreamSynchronize(x5));
  int x21 = 0;
  while (x21 != 33554432) {
    CUDA_CALL(hipMemcpy(x9, x15[x21], 33554432, hipMemcpyDeviceToHost));
    if (x9[0] != 1) printf("error");
    x21 = x21 + 1;
  }
  CUDA_CALL(hipFree(x11));
  CUDA_CALL(hipFree(x15));
  NCCLCHECK(ncclCommDestroy(x7));
  MPICHECK(MPI_Finalize());
  printf("[MPI Rank %d] Success \n", x1);
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
