/*****************************************
Emitting C Generated Code
*******************************************/
#include "nccl_header.h"
#include <string.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include "mpi_header.h"
/**************** Snippet ****************/
void Snippet(int x0) {
  int x1 = 0;
  int x2 = 0;
  MPICHECK(MPI_Init(NULL, NULL));
  int x3 = MPI_Comm_rank(MPI_COMM_WORLD, &x1);
  MPICHECK(x3);
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &x2));
  printf("myRank: %d, nRanks: %d\n", x1, x2);
  ncclUniqueId x4;
  ncclComm_t x5;
  hipStream_t x6;
  if (x1 == 0) NCCLCHECK(ncclGetUniqueId(&x4));
  MPICHECK(MPI_Bcast(&x4, NCCL_UNIQUE_ID_BYTES, MPI_BYTE, 0, MPI_COMM_WORLD));
  float* x7 = (float*)malloc(1024 * sizeof(float));
  int x8 = 0;
  while (x8 != 1024) {
    x7[x8] = 2.0;
    x8 = x8 + 1;
  }
  CUDA_CALL(hipSetDevice(x1));
  float** x9 = (float**)malloc(x2 * sizeof(float*));
  float** x10 = (float**)malloc(x2 * sizeof(float*));
  int x11 = x2;
  int x12 = 0;
  while (x12 != x11) {
    int x13 = x12;
    CUDA_CALL(hipMalloc(&x9[x13], (size_t)(1024 * sizeof(float))));
    CUDA_CALL(hipMalloc(&x10[x13], (size_t)(1024 * sizeof(float))));
    CUDA_CALL(hipMemcpy(x9[x13], x7, (size_t)(1024 * sizeof(float)), hipMemcpyHostToDevice));
    x12 = x12 + 1;
  }
  CUDA_CALL(hipStreamCreateWithFlags(&x6, hipStreamDefault));
  NCCLCHECK(ncclCommInitRank(&x5, x2, x4, x1));
  int x14 = x2;
  ncclDataType_t x15 = ncclFloat;
  NCCLCHECK(ncclGroupStart());
  int x16 = 0;
  while (x16 != x14) {
    int x17 = x16;
    NCCLCHECK(ncclSend(x9[x17], 1024, x15, x17, x5, x6));
    NCCLCHECK(ncclRecv(x10[x17], 1024, x15, x17, x5, x6));
    x16 = x16 + 1;
  }
  NCCLCHECK(ncclGroupEnd());
  CUDA_CALL(hipStreamSynchronize(x6));
  int x18 = 0;
  int x19 = x2;
  int x20 = 0;
  while (x20 != x19) {
    float* x21 = (float*)malloc(1024 * sizeof(float));
    CUDA_CALL(hipMemcpy(x21, x10[x20], (size_t)(1024 * sizeof(float)), hipMemcpyDeviceToHost));
    int x22 = 0;
    while (x22 != 1024) {
      if (x21[x22] != 2) x18 = x18 + 1;
      x22 = x22 + 1;
    }
    x20 = x20 + 1;
  }
  int x23 = x2;
  int x24 = 0;
  while (x24 != x23) {
    int x25 = x24;
    CUDA_CALL(hipFree(x9[x25]));
    CUDA_CALL(hipFree(x10[x25]));
    x24 = x24 + 1;
  }
  NCCLCHECK(ncclCommDestroy(x5));
  MPICHECK(MPI_Finalize());
  if (x18 != 0) printf("[MPI Rank %d] Found %d errors.\n", x1, x18);
  else printf("[MPI Rank %d] Success \n", x1);
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
