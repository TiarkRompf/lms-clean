#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#include <string.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
/************* Functions **************/
__global__ void x7(int* x8, int* x9, int x10, int x11) {
  // this is the transpose kernel
  // arg0: 2D Input Matrix (dimY x dimX) may not be multiples of 32
  // arg1: 2D Output Matrix (dimX x dimY)
  // arg2: dimY of input
  // arg3: dimX of input
  // caller must use <<<dim3((dimX+31)/32, (dimY+31)/32, 1), dim3(32, 8, 1)>>>
  // using gridDimX=(dimX+31)/32, gridDimY=(dimY+31)/32, blockDimX=32, blockDimY=8
  __shared__ int x12[1056];
  // read data from input array to shared memory
  int x13 = 0;
  while (x13 < 32) {
    int x14 = x13;
    int x15 = blockIdx.y * 32 + (threadIdx.y + x14);
    if (x15 < x10 && blockIdx.x * 32 + threadIdx.x < x11) x12[33 * (threadIdx.y + x14) + threadIdx.x] = x8[x15 * x11 + (blockIdx.x * 32 + threadIdx.x)];
    x13 = x13 + 8;
  }
  // sync threads
  __syncthreads();
  // write data from shared memory to output array
  int x16 = 0;
  while (x16 < 32) {
    int x17 = x16;
    int x18 = blockIdx.x * 32 + (threadIdx.y + x17);
    if (x18 < x11 && blockIdx.y * 32 + threadIdx.x < x10) x9[x18 * x10 + (blockIdx.y * 32 + threadIdx.x)] = x12[33 * threadIdx.x + (threadIdx.y + x17)];
    x16 = x16 + 8;
  }
}
/**************** Snippet ****************/
void Snippet(int x0) {
  int* x1 = (int*)malloc(20000 * sizeof(int));
  int* x2 = (int*)malloc(20000 * sizeof(int));
  int x3 = 0;
  while (x3 != 20000) {
    int x4 = x3;
    x1[x4] = x4 + 1;
    x3 = x3 + 1;
  }
  int* x5 = (int*)malloc(0 * sizeof(int));
  CUDA_CALL(hipMalloc(&x5, (size_t)(20000 * sizeof(int))));
  int* x6 = (int*)malloc(0 * sizeof(int));
  CUDA_CALL(hipMalloc(&x6, (size_t)(20000 * sizeof(int))));
  CUDA_CALL(hipMemcpy(x5, x1, (size_t)(20000 * sizeof(int)), hipMemcpyHostToDevice));
  x7<<<dim3(7, 4, 1), dim3(32, 8, 1)>>>(x5, x6, 100, 200);
  CUDA_CALL(hipMemcpy(x2, x6, (size_t)(20000 * sizeof(int)), hipMemcpyDeviceToHost));
  int x19 = 0;
  while (x19 != 200) {
    int x20 = x19;
    int x21 = 0;
    int x22 = 100 * x20;
    while (x21 != 100) {
      int x23 = x21;
      if (x1[200 * x23 + x20] != x2[x22 + x23]) {
        printf("Transpose Incorrect!\n");
        fflush(stdout); fflush(stderr); exit(1);
      }
      x21 = x21 + 1;
    }
    x19 = x19 + 1;
  }
  printf("Transpose Correct\n");
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
