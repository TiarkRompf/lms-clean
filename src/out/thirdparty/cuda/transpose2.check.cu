#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#include <string.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
/************* Functions **************/
__global__ void x7(int* x8, int* x9, int x10, int x11) {
  // this is the transpose kernel
  // arg0: 2D Input Matrix (dimY x dimX) where dimY and dimX are multiples of 32
  // arg1: 2D Output Matrix (dimX x dimY)
  // caller must use <<<dim3(dimX/32, dimY/32, 1), dim3(32, 32, 1)>>>
  // using gridDimX=dimX/32, gridDimY=dimY/32, blockDimX=32, blockDimY=32
  __shared__ int x12[1056];
  // read data from input array to shared memory
  x12[33 * threadIdx.y + threadIdx.x] = x8[(blockIdx.y * 32 + threadIdx.y) * x11 + (blockIdx.x * 32 + threadIdx.x)];
  // sync threads
  __syncthreads();
  // write date from shared memory to output array
  x9[(blockIdx.x * 32 + threadIdx.y) * x10 + (blockIdx.y * 32 + threadIdx.x)] = x12[33 * threadIdx.x + threadIdx.y];
}
/**************** Snippet ****************/
void Snippet(int x0) {
  int* x1 = (int*)malloc(8192 * sizeof(int));
  int* x2 = (int*)malloc(8192 * sizeof(int));
  int x3 = 0;
  while (x3 != 8192) {
    int x4 = x3;
    x1[x4] = x4 + 1;
    x3 = x3 + 1;
  }
  int* x5 = (int*)malloc(0 * sizeof(int));
  CUDA_CALL(hipMalloc(&x5, (size_t)(8192 * sizeof(int))));
  int* x6 = (int*)malloc(0 * sizeof(int));
  CUDA_CALL(hipMalloc(&x6, (size_t)(8192 * sizeof(int))));
  CUDA_CALL(hipMemcpy(x5, x1, (size_t)(8192 * sizeof(int)), hipMemcpyHostToDevice));
  x7<<<dim3(4, 2, 1), dim3(32, 32, 1)>>>(x5, x6, 64, 128);
  CUDA_CALL(hipMemcpy(x2, x6, (size_t)(8192 * sizeof(int)), hipMemcpyDeviceToHost));
  int x13 = 0;
  while (x13 != 128) {
    int x14 = x13;
    int x15 = 0;
    int x16 = 64 * x14;
    while (x15 != 64) {
      int x17 = x15;
      if (x1[128 * x17 + x14] != x2[x16 + x17]) {
        printf("Transpose Incorrect!\n");
        fflush(stdout); fflush(stderr); exit(1);
      }
      x15 = x15 + 1;
    }
    x13 = x13 + 1;
  }
  printf("Transpose Correct\n");
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
