#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif
#include <string.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include "scanner_header.h"
/************* Functions **************/
__global__ void x7(float* x8, int x9, float* x10, int x11, float* x12, int x13) {
  // this is cuda concat kernel.
  // It concatenates two 3D arrays and concat on the innermost dimension (dim2).
  // arg0: first input array
  // arg1: dim2 of first input array
  // arg2: second input array
  // arg3: dim2 of second input array
  // arg4: output array
  // arg5: product of other two dimensions (dim0 * dim1)
  // call constraint: arg1 + arg3 = out.dim2
  int x14 = blockIdx.x * blockDim.x + threadIdx.x;
  int x15 = x9 + x11;
  if (x14 < x13 * x15) {
    int x16 = x14 % x15;
    x12[x14] = x16 < x9 ? x8[x14 / x15 * x9 + x16] : x10[x14 / x15 * x11 + (x16 - x9)];
  }
}
/**************** Snippet ****************/
void Snippet(int x0) {
  float* x1 = (float*)malloc(18 * sizeof(float));
  scan_float("golden/concat2/input0.data", x1, 18);
  float* x2 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x2, (size_t)(18 * sizeof(float))));
  CUDA_CALL(hipMemcpy(x2, x1, (size_t)(18 * sizeof(float)), hipMemcpyHostToDevice));
  float* x3 = (float*)malloc(30 * sizeof(float));
  scan_float("golden/concat2/input1.data", x3, 30);
  float* x4 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x4, (size_t)(30 * sizeof(float))));
  CUDA_CALL(hipMemcpy(x4, x3, (size_t)(30 * sizeof(float)), hipMemcpyHostToDevice));
  float* x5 = (float*)malloc(48 * sizeof(float));
  float* x6 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x6, (size_t)(48 * sizeof(float))));
  x7<<<dim3(1, 1, 1), dim3(512, 1, 1)>>>(x2, 3, x4, 5, x6, 6);
  CUDA_CALL(hipMemcpy(x5, x6, (size_t)(48 * sizeof(float)), hipMemcpyDeviceToHost));
  check_float_array("golden/concat2/output.data", x5, 48);
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
