#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif
#include <string.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include "scanner_header.h"
/************* Functions **************/
__global__ void x9(float** x10, float* x11) {
  // this is cuda 2-section concat kernel for 3D inputs at axis 2.
  // It concatenates 2 3D arrays on the innermost dimension (dim2).
  // arg0: array of input input arrays
  // arg1: output array
  // call constraint: in.size = 2
  // call constraint: sum of in(i).size = out.size for i in [0, 2)
  int x12 = blockIdx.x * blockDim.x + threadIdx.x;
  if (x12 < 48) {
    int x13 = x12 % 8;
    if (x13 < 3) x11[x12] = x10[0][x12 / 8 * 3 + x13];
    else x11[x12] = x10[1][x12 / 8 * 5 + (x13 - 3)];
  }
}
/**************** Snippet ****************/
void Snippet(int x0) {
  float* x1 = (float*)malloc(18 * sizeof(float));
  scan_float("golden/concat2/input0.data", x1, 18);
  float* x2 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x2, (size_t)(18 * sizeof(float))));
  CUDA_CALL(hipMemcpy(x2, x1, (size_t)(18 * sizeof(float)), hipMemcpyHostToDevice));
  float* x3 = (float*)malloc(30 * sizeof(float));
  scan_float("golden/concat2/input1.data", x3, 30);
  float* x4 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x4, (size_t)(30 * sizeof(float))));
  CUDA_CALL(hipMemcpy(x4, x3, (size_t)(30 * sizeof(float)), hipMemcpyHostToDevice));
  float* x5 = (float*)malloc(48 * sizeof(float));
  float* x6 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x6, (size_t)(48 * sizeof(float))));
  float** x7 = (float**)malloc(2 * sizeof(float*));
  x7[0] = x2;
  x7[1] = x4;
  float** x8 = (float**)malloc(0 * sizeof(float*));
  CUDA_CALL(hipMalloc(&x8, (size_t)(2 * sizeof(float*))));
  CUDA_CALL(hipMemcpy(x8, x7, (size_t)(2 * sizeof(float*)), hipMemcpyHostToDevice));
  x9<<<dim3(1, 1, 1), dim3(512, 1, 1)>>>(x8, x6);
  CUDA_CALL(hipMemcpy(x5, x6, (size_t)(48 * sizeof(float)), hipMemcpyDeviceToHost));
  // check general cuda3DConcat kernel
  check_float_array("golden/concat2/output.data", x5, 48);
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
