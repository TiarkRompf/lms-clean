/*************************************************************************
 * Copyright (c) 2015-2016, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENCE.txt for license information
 ************************************************************************/

#include <stdio.h>
#include <sys/types.h>
#include <unistd.h>

#include "mpi.h"
#include "nccl.h"
#include "test_utilities.h"

#define SIZE 128
#define NITERS 1

int main(int argc, char *argv[]) {
  ncclUniqueId commId;
  int size, rank;
  ncclResult_t ret;

  MPI_Init(&argc, &argv);
  MPI_Comm_size(MPI_COMM_WORLD, &size);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);

  if (argc < size) {
    if (rank == 0)
      printf("Usage : %s <GPU list per rank>\n", argv[0]);
    exit(1);
  }

  int gpu = atoi(argv[rank + 1]);

  // We have to set our device before NCCL init
  CUDACHECK(hipSetDevice(gpu));
  MPI_Barrier(MPI_COMM_WORLD);

  // NCCL Communicator creation
  ncclComm_t comm;
  NCCLCHECK(ncclGetUniqueId(&commId));
  MPI_Bcast(&commId, NCCL_UNIQUE_ID_BYTES, MPI_CHAR, 0, MPI_COMM_WORLD);
  ret = ncclCommInitRank(&comm, size, commId, rank);
  if (ret != ncclSuccess) {
    printf("NCCL Init failed (%d) '%s'\n", ret, ncclGetErrorString(ret));
    exit(1);
  }

  // CUDA stream creation
  hipStream_t stream;
  CUDACHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  // Initialize input values
  int *dptr;
  CUDACHECK(hipMalloc(&dptr, SIZE * 2 * sizeof(int)));
  int *val = (int *)malloc(SIZE * sizeof(int));
  for (int v = 0; v < SIZE; v++) {
    val[v] = rank + 1;
  }
  CUDACHECK(hipMemcpy(dptr, val, SIZE * sizeof(int), hipMemcpyHostToDevice));

  // Compute final value
  int ref = size * (size + 1) / 2;

  // Run allreduce
  int errors = 0;
  for (int i = 0; i < NITERS; i++) {
    NCCLCHECK(ncclAllReduce((const void *)dptr, (void *)(dptr + SIZE), SIZE,
                            ncclInt, ncclSum, comm, stream));
  }

  // Check results
  CUDACHECK(hipStreamSynchronize(stream));
  CUDACHECK(hipMemcpy(val, (dptr + SIZE), SIZE * sizeof(int),
                       hipMemcpyDeviceToHost));
  for (int v = 0; v < SIZE; v++) {
    if (val[v] != ref) {
      errors++;
      printf("[%d] Error at %d : got %d instead of %d\n", rank, v, val[v], ref);
    }
  }
  CUDACHECK(hipFree(dptr));

  MPI_Allreduce(MPI_IN_PLACE, &errors, 1, MPI_INTEGER, MPI_SUM, MPI_COMM_WORLD);
  if (rank == 0) {
    if (errors)
      printf("%d errors. Test FAILED.\n", errors);
    else
      printf("Test PASSED.\n");
  }

  MPI_Finalize();
  ncclCommDestroy(comm);
  return errors ? 1 : 0;
}